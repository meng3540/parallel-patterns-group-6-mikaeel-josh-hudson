#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// Compute sum reduction
__global__ void SimpleSumReductionKernel(float* input, float* output) {
    unsigned int i = threadIdx.x;
    // Perform reduction in shared memory
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        if (threadIdx.x % stride == 0) {
            input[i] += input[i + stride];
        }
        __syncthreads(); // Ensure all threads have completed the current stride
    }
    // Write the result of the reduction to the output
    if (threadIdx.x == 0) {
        *output = input[0];
    }
}

// Function to print a matrix
void printMatrix(float* matrix, int numRows, int numColumns, const char* name) {
    printf("\nMatrix %s:\n", name);
    for (int i = 0; i < numRows; i++) {
        for (int j = 0; j < numColumns; j++) {
            printf("%.1f ", matrix[i * numColumns + j]);
        }
        printf("\n");
    }
}

int main(int argc, char** argv)
{
    float* hostA; // The A matrix on the host
    float* hostC; // The output C matrix on the host
    float* deviceA; // Device copy of A matrix
    float* deviceC; // Device result of sum reduction
    int numARows = 1; // Number of rows in the matrix A
    int numAColumns = 64; // Number of columns in the matrix A
    int numCRows = 1; // Number of rows in the matrix C
    int numCColumns = 1; // Number of columns in the matrix C

    // Calculate the size in bytes for matrix A and C
    int sizeA = (numARows * numAColumns) * sizeof(float);
    int sizeC = (numCRows * numCColumns) * sizeof(float);

    // Allocate and initialize the A matrix on the host
    hostA = (float*)malloc(sizeA);
    for (int i = 0; i < numARows; i++) {
        for (int j = 0; j < numAColumns; j++) {
            hostA[i * numAColumns + j] = j + 1; // Initialize with values 1, 2, 3, ..., 64
        }
    }

    // Allocate the C matrix on the host
    hostC = (float*)malloc(sizeC);

    // Print the A matrix
    printMatrix(hostA, numARows, numAColumns, "A");

    printf("\nThe dimensions of A are %d x %d\n", numARows, numAColumns);
    printf("The dimensions of C are %d x %d\n", numCRows, numCColumns);

    // Allocate memory on the GPU for matrices A and C
    hipMalloc((void**)&deviceA, sizeA);
    hipMalloc((void**)&deviceC, sizeof(float));

    // Copy the A matrix from the host to the device
    hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);

    // Initialize the block dimensions
    int blockSize = numAColumns; // Number of threads per block
    int gridSize = (numARows * numAColumns + blockSize - 1) / blockSize; // Number of blocks

    printf("The block dimensions are %d\n", blockSize);
    printf("The grid dimensions are %d\n", gridSize);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    // Launch the GPU Kernel
    SimpleSumReductionKernel << <gridSize, blockSize >> > (deviceA, deviceC);
    hipDeviceSynchronize();

    // Record the stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel execution time: %f ms\n", elapsedTime);

    // Calculate memory bandwidth
    int totalBytes = sizeA + sizeC;
    float bandwidth = (totalBytes) / (elapsedTime * 1e3); // Bytes/s
    printf("Effective memory bandwidth: %f bytes/s\n", bandwidth);

    // Copy the result from the device back to the host
    float result;
    hipMemcpy(&result, deviceC, sizeof(float), hipMemcpyDeviceToHost);

    // Free the GPU memory
    hipFree(deviceA);
    hipFree(deviceC);

    // Print the result
    printf("\nResult: %f\n", result);

    // Free host memory
    free(hostA);
    free(hostC);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
